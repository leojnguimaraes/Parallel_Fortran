
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloworld() {
  printf("Hello, World!\n");
}

int main() {
  helloworld<<<1,1>>>();
  hipDeviceSynchronize();
  return 0;
}

